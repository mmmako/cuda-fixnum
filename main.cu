#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <cassert>
#include <vector>

#include "fixnum/warp_fixnum.cu"
#include "array/fixnum_array.h"
#include "functions/modexp.cu"
#include "functions/multi_modexp.cu"
#include "modnum/modnum_monty_redc.cu"
#include "modnum/modnum_monty_cios.cu"

const unsigned int bytes_per_elem = 128;
const unsigned int io_bytes_per_elem = 96;


using namespace std;
using namespace cuFIXNUM;

template< typename fixnum, typename modnum>
__device__ void mul_quad(fixnum &r0, fixnum &r1, fixnum a0, fixnum a1, fixnum b0, fixnum b1, modnum mod, fixnum non_residue) {
    //fixnum a0b0, a1b1, a1b1nr;

    //mod.mul(a0b0, a0, b0);
    //mod.mul(a1b1, a1, b1);
    //mod.mul(a1b1nr, a1b1, non_residue);

    //fixnum a0pb0, a1pb1;
    //mod.add(a0pb0, a0, b0);
    //mod.add(a1pb1, a1, b1);

    //fixnum prod, sub1;
    //mod.mul(prod, a0pb0, a1pb1);
    //mod.sub(sub1, prod, a0b0);

    fixnum a0b0, a0b1, a1b0, a1b1, a1b1nr;
    mod.mul(a0b0, a0, b0);
    mod.mul(a0b1, a0, b1);
    mod.mul(a1b0, a1, b0);
    mod.mul(a1b1, a1, b1);
    mod.mul(a1b1nr, a1b1, non_residue);

    fixnum s0, s1;
    //mod.add(s0, a0b0, a1b1nr);
    //mod.sub(s1, sub1, a1b1);
    mod.add(s0, a0b0, a1b1nr);
    mod.add(s1, a0b1, a1b0);

    r0 = s0;
    r1 = s1;
}

template< typename fixnum >
struct mul_quad_and_convert {
  // redc may be worth trying over cios
  typedef modnum_monty_cios<fixnum> modnum;
  __device__ void operator()(fixnum &r0, fixnum &r1, fixnum a0, fixnum a1, fixnum b0, fixnum b1, fixnum my_mod, fixnum non_residue) {
      modnum mod = modnum(my_mod);

      fixnum sm0, sm1;

      fixnum am0, am1;
      fixnum bm0, bm1;
      fixnum non_residuem;
      mod.to_modnum(am0, a0);
      mod.to_modnum(am1, a1);
      mod.to_modnum(bm0, b0);
      mod.to_modnum(bm1, b1);
      mod.to_modnum(non_residuem, non_residue);
      
      mul_quad(sm0, sm1, am0, am1, bm0, bm1, mod, non_residuem);

      fixnum s0, s1;
      mod.from_modnum(s0, sm0);
      mod.from_modnum(s1, sm1);

      r0 = s0;
      r1 = s1;
  }
};

template< int fn_bytes, typename fixnum_array >
void print_fixnum_array(fixnum_array* res, int nelts) {
    int lrl = fn_bytes*nelts;
    uint8_t local_results[lrl];
    int ret_nelts;
    for (int i = 0; i < lrl; i++) {
      local_results[i] = 0;
    }
    res->retrieve_all(local_results, fn_bytes*nelts, &ret_nelts);

    for (int i = 0; i < lrl; i++) {
      printf("%i ", local_results[i]);
    }
    printf("\n");
}

template< int fn_bytes, typename fixnum_array >
vector<uint8_t*> get_fixnum_array(fixnum_array* res, int nelts) {
    int lrl = fn_bytes*nelts;
    //uint8_t local_results[lrl];
    uint8_t* local_results = new uint8_t[lrl];
    int ret_nelts;
    for (int i = 0; i < lrl; i++) {
      local_results[i] = 0;
    }
    res->retrieve_all(local_results, fn_bytes*nelts, &ret_nelts);
    vector<uint8_t*> res_v;
    for (int n = 0; n < nelts; n++) {
      uint8_t* a = (uint8_t*)malloc(fn_bytes*sizeof(uint8_t));
      for (int i = 0; i < fn_bytes; i++) {
        a[i] = local_results[n*fn_bytes + i];
      }
      res_v.emplace_back(a);
    }
    delete[](local_results);
    return res_v;
}


template< int fn_bytes, typename word_fixnum, template <typename> class Func >
std::pair<std::vector<uint8_t*>, std::vector<uint8_t*>> compute_product(std::vector<uint8_t*> a0, std::vector<uint8_t*> a1,
        std::vector<uint8_t*> b0, std::vector<uint8_t*> b1, uint8_t* input_m_base, uint8_t* non_residue) {
    typedef warp_fixnum<fn_bytes, word_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;

    int nelts = a0.size();

    uint8_t *input_a0 = new uint8_t[fn_bytes * nelts];
    uint8_t *input_a1 = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i) {
      input_a0[i] = a0[i/fn_bytes][i%fn_bytes];
      input_a1[i] = a1[i/fn_bytes][i%fn_bytes];
    }

    uint8_t *input_b0 = new uint8_t[fn_bytes * nelts];
    uint8_t *input_b1 = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i) {
      input_b0[i] = b0[i/fn_bytes][i%fn_bytes];
      input_b1[i] = b1[i/fn_bytes][i%fn_bytes];
    }

    uint8_t *input_m = new uint8_t[fn_bytes * nelts];
    uint8_t *input_nr = new uint8_t[fn_bytes * nelts];
    for (int i = 0; i < fn_bytes * nelts; ++i) {
      input_m[i] = input_m_base[i%fn_bytes];
      input_nr[i] = non_residue[i%fn_bytes];
    }

    // TODO reuse modulus as a constant instead of passing in nelts times
    fixnum_array *res0, *res1, *in_a0, *in_a1, *in_b0, *in_b1, *inM, *inNR;
    in_a0 = fixnum_array::create(input_a0, fn_bytes * nelts, fn_bytes);
    in_a1 = fixnum_array::create(input_a1, fn_bytes * nelts, fn_bytes);
    in_b0 = fixnum_array::create(input_b0, fn_bytes * nelts, fn_bytes);
    in_b1 = fixnum_array::create(input_b1, fn_bytes * nelts, fn_bytes);
    inM = fixnum_array::create(input_m, fn_bytes * nelts, fn_bytes);
    inNR = fixnum_array::create(input_nr, fn_bytes * nelts, fn_bytes);
    res0 = fixnum_array::create(nelts);
    res1 = fixnum_array::create(nelts);

    fixnum_array::template map<Func>(res0, res1, in_a0, in_a1, in_b0, in_b1, inM, inNR);

    vector<uint8_t*> v_res0 = get_fixnum_array<fn_bytes, fixnum_array>(res0, nelts);
    vector<uint8_t*> v_res1 = get_fixnum_array<fn_bytes, fixnum_array>(res1, nelts);

    //TODO to do stage 1 field arithmetic, instead of a map, do a reduce

    delete in_a0;
    delete in_a1;
    delete in_b0;
    delete in_b1;
    delete inM;
    delete res0;
    delete res1;
    delete[] input_a0;
    delete[] input_a1;
    delete[] input_b0;
    delete[] input_b1;
    delete[] input_m;
    return std::make_pair(v_res0, v_res1);
}

uint8_t* read_mnt_fq(FILE* inputs) {
  uint8_t* buf = (uint8_t*)calloc(bytes_per_elem, sizeof(uint8_t));
  // the input is montgomery representation x * 2^768 whereas cuda-fixnum expects x * 2^1024 so we shift over by (1024-768)/8 bytes
  fread((void*)(buf), io_bytes_per_elem*sizeof(uint8_t), 1, inputs);
  return buf;
}

void write_mnt_fq(uint8_t* fq, FILE* outputs) {
  fwrite((void *) fq, io_bytes_per_elem * sizeof(uint8_t), 1, outputs);
}

void print_array(uint8_t* a) {
  for (int j = 0; j < 128; j++) {
    printf("%x ", ((uint8_t*)(a))[j]);
  }
  printf("\n");
}

int main(int argc, char* argv[]) {
  setbuf(stdout, NULL);

  // mnt4_q
  uint8_t mnt4_modulus[bytes_per_elem] = {1,128,94,36,222,99,144,94,159,17,221,44,82,84,157,227,240,37,196,154,113,16,136,99,164,84,114,118,233,204,90,104,56,126,83,203,165,13,15,184,157,5,24,242,118,231,23,177,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  uint8_t non_residue[bytes_per_elem] = {13,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  // mnt6_q
  //uint8_t mnt6_modulus[bytes_per_elem] = {1,0,0,64,226,118,7,217,79,58,161,15,23,153,160,78,151,87,0,63,188,129,195,214,164,58,153,52,118,249,223,185,54,38,33,41,148,202,235,62,155,169,89,200,40,92,108,178,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  auto inputs = fopen(argv[2], "r");
  auto outputs = fopen(argv[3], "w");

  size_t n;

   while (true) {
    size_t elts_read = fread((void *) &n, sizeof(size_t), 1, inputs);
    if (elts_read == 0) { break; }

    std::vector<uint8_t*> x0;
    std::vector<uint8_t*> x1;
    for (size_t i = 0; i < n; ++i) {
      x0.emplace_back(read_mnt_fq(inputs));
      x1.emplace_back(read_mnt_fq(inputs));
    }

    std::vector<uint8_t*> y0;
    std::vector<uint8_t*> y1;
    for (size_t i = 0; i < n; ++i) {
      y0.emplace_back(read_mnt_fq(inputs));
      y1.emplace_back(read_mnt_fq(inputs));
    }

    std::pair<std::vector<uint8_t*>, std::vector<uint8_t*>> res_x
        = compute_product<bytes_per_elem, u64_fixnum, mul_quad_and_convert>(x0, x1, y0, y1, mnt4_modulus, non_residue);

    for (size_t i = 0; i < n; ++i) {
      write_mnt_fq(res_x.first[i], outputs);
      write_mnt_fq(res_x.second[i], outputs);
    }

    for (size_t i = 0; i < n; ++i) {
      free(x0[i]);
      free(x1[i]);
      free(y0[i]);
      free(y1[i]);
      free(res_x.first[i]);
      free(res_x.second[i]);
    }
  }

  return 0;
}

